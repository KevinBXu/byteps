#include "hip/hip_runtime.h"
#include "terngrad.cuh"
#include "math.h"
#include <stdio.h>
#include <iostream>

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void find_grad_max(const void* gpu_ptr, size_t len, float* result){
    float* ptr = reinterpret_cast<float*>(const_cast<void*>(gpu_ptr));
    float grad_max;
    if (ptr[0] >= 0) grad_max = ptr[0];
    else grad_max = -ptr[0];
    float grad_abs;
    for(size_t i = 0; i < len; i++){
        if (ptr[i] >= 0) grad_abs = ptr[i];
        else grad_abs = -ptr[i];
        if (grad_abs > grad_max) grad_max = grad_abs;
    }
    *result = grad_max;
}

__global__ void terngrad_compress_kernel(const void* gpu_ptr, size_t len, hiprandState *state, float grad_max){
    //threadIdx.x contains the index of the current thread within its block, 
    //and blockDim.x contains the number of threads in the block
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float* ptr = reinterpret_cast<float*>(const_cast<void*>(gpu_ptr));
    float x;
    int index = threadIdx.x;
    int stride = blockDim.x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    for(size_t i = index; i < len; i+=stride) {
        x = hiprand_uniform(&localState);
        if(x < fabsf(ptr[i])/grad_max) {
            if (ptr[i] > 0) ptr[i] = 1.0;
            else ptr[i] = -1.0;
        }
        else ptr[i] = 0.0;
        printf("Done index %d\n", i);
    }
    /* Copy state back to global memory */
    state[id] = localState;
}

void terngrad_compress(const void* gpu_ptr, size_t len){
    float* ptr = reinterpret_cast<float*>(const_cast<void*>(gpu_ptr));
    float grad_max;
    float* grad_max_answer;
    hipMalloc(&grad_max_answer, sizeof(float));
    find_grad_max<<<1, 1>>>(gpu_ptr, len, grad_max_answer);
    hipMemcpy(&grad_max, grad_max_answer, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(grad_max_answer);
    std::cout << "grad_max: " << grad_max << std::endl;
    
    const unsigned int threadsPerBlock = 64;
    // TODO: first try one block, then increase block number
    const unsigned int blockCount = 1;
    //const unsigned int blockCount = (len + threadsPerBlock - 1) / threadsPerBlock;
    const unsigned int totalThreads = threadsPerBlock * blockCount;
    hiprandState *devStates;
    // /* Allocate space for results on host */
    // hostResults = (unsigned int *)calloc(totalThreads, sizeof(int));
    /* Allocate space for prng states on device */
    hipMalloc((void**)&devStates, totalThreads * sizeof(hiprandState));
    std::cout << "Done mallocing for devStates" << std::endl;
    /* Setup prng states */
    setup_kernel<<<blockCount, threadsPerBlock>>>(devStates);
    std::cout << "Done setup" << std::endl;
    terngrad_compress_kernel<<<blockCount, threadsPerBlock>>>(gpu_ptr, len, devStates, grad_max);
    /* Cleanup */
    hipFree(devStates);
}

void terngrad_decompress(const void* gpu_ptr, float scale, size_t len){
    // TODO: time the gradient with a scale
    // For now, just do nothing as I haven't figured out where to put scale
}
