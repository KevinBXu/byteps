#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "test.cuh"
//#include <iostream>

__global__ void test_kernel(void){}

void test_wrapper(void){
    test_kernel <<<1, 1>>> ();
    std::cout << "Tested CUDA kernel" << std::endl;
}
